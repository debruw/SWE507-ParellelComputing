#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Structure for BMP Header
#pragma pack(push, 1)
typedef struct {
	unsigned short bfType;
	unsigned int bfSize;
	unsigned short bfReserved1;
	unsigned short bfReserved2;
	unsigned int bfOffBits;
} BITMAPFILEHEADER;

typedef struct {
	unsigned int biSize;
	int biWidth;
	int biHeight;
	unsigned short biPlanes;
	unsigned short biBitCount;
	unsigned int biCompression;
	unsigned int biSizeImage;
	int biXPelsPerMeter;
	int biYPelsPerMeter;
	unsigned int biClrUsed;
	unsigned int biClrImportant;
} BITMAPINFOHEADER;
#pragma pack(pop)

unsigned char* image;
int width, height;

// Constant memory for kernel
__constant__ float d_kernel_const[3][3];  // For 3x3 kernels

// Function to load a BMP image
int load_bmp(const char* filename) {
	FILE* file = fopen(filename, "rb");
	if (!file) {
		printf("Error: Failed to open BMP file.\n");
		return 0;
	}

	BITMAPFILEHEADER fileHeader;
	BITMAPINFOHEADER infoHeader;

	fread(&fileHeader, sizeof(BITMAPFILEHEADER), 1, file);
	fread(&infoHeader, sizeof(BITMAPINFOHEADER), 1, file);

	width = infoHeader.biWidth;
	height = infoHeader.biHeight;

	image = (unsigned char*)malloc(width * height * 3);
	if (!image) {
		printf("Error: Failed to allocate memory for image.\n");
		fclose(file);
		return 0;
	}

	fseek(file, fileHeader.bfOffBits, SEEK_SET);
	fread(image, sizeof(unsigned char), width * height * 3, file);
	fclose(file);
	return 1;
}

// Function to save a BMP image
void save_bmp(const char* filename, unsigned char* imageData) {
	FILE* file = fopen(filename, "wb");
	if (!file) {
		printf("Error: Failed to save BMP file.\n");
		return;
	}

	BITMAPFILEHEADER fileHeader;
	BITMAPINFOHEADER infoHeader;

	fileHeader.bfType = 0x4D42;
	fileHeader.bfSize = sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER) + width * height * 3;
	fileHeader.bfReserved1 = fileHeader.bfReserved2 = 0;
	fileHeader.bfOffBits = sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER);

	infoHeader.biSize = sizeof(BITMAPINFOHEADER);
	infoHeader.biWidth = width;
	infoHeader.biHeight = height;
	infoHeader.biPlanes = 1;
	infoHeader.biBitCount = 24;
	infoHeader.biCompression = 0;
	infoHeader.biSizeImage = width * height * 3;
	infoHeader.biXPelsPerMeter = 0;
	infoHeader.biYPelsPerMeter = 0;
	infoHeader.biClrUsed = 0;
	infoHeader.biClrImportant = 0;

	fwrite(&fileHeader, sizeof(BITMAPFILEHEADER), 1, file);
	fwrite(&infoHeader, sizeof(BITMAPINFOHEADER), 1, file);
	fwrite(imageData, sizeof(unsigned char), width * height * 3, file);

	fclose(file);
}

// Clamp function on device
__device__ int clamp(int val, int min, int max) {
	if (val < min) return min;
	if (val > max) return max;
	return val;
}

// CUDA kernel using constant memory
__global__ void d_applyConvolutionKernel(unsigned char* d_input, unsigned char* d_output, int imageWidth, int imageHeight, int kernelSize) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int kernelRadius = kernelSize / 2;

	if (x < imageWidth && y < imageHeight) {
		float valueR = 0.0f, valueG = 0.0f, valueB = 0.0f;

		for (int ky = -kernelRadius; ky <= kernelRadius; ++ky) {
			for (int kx = -kernelRadius; kx <= kernelRadius; ++kx) {
				int imageX = x + kx;
				int imageY = y + ky;
				int kernelX = kx + kernelRadius;
				int kernelY = ky + kernelRadius;

				float kernelVal = d_kernel_const[kernelY][kernelX];

				if (imageX < 0 || imageX >= imageWidth || imageY < 0 || imageY >= imageHeight) {
					valueR += 0.0f * kernelVal;
					valueG += 0.0f * kernelVal;
					valueB += 0.0f * kernelVal;
				}
				else {
					int idx = (imageY * imageWidth + imageX) * 3;
					valueR += d_input[idx + 2] * kernelVal;
					valueG += d_input[idx + 1] * kernelVal;
					valueB += d_input[idx + 0] * kernelVal;
				}
			}
		}

		int outputIdx = (y * imageWidth + x) * 3;
		d_output[outputIdx + 2] = clamp(int(valueR), 0, 255);
		d_output[outputIdx + 1] = clamp(int(valueG), 0, 255);
		d_output[outputIdx + 0] = clamp(int(valueB), 0, 255);
	}
}

int main() {
	// Define a 3x3 box blur kernel
	float kernel[3][3] = {
		{1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f},
		{1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f},
		{1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f}
	};

	/*float kernel[3][3] = {
	{0.0f, -1.0f, 0.0f},
	{-1.0f, 5.0f, -1.0f},
	{0.0f, -1.0f, 0.0f}
	};*/
	int kernelSize = 3;

	if (!load_bmp("lena.bmp")) {
		return 1;
	}

	// Copy kernel to constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_const), kernel, sizeof(float) * 3 * 3);

	unsigned char* d_inputImage, * d_outputImage;
	size_t imageSize = width * height * 3;

	hipMalloc(&d_inputImage, imageSize);
	hipMalloc(&d_outputImage, imageSize);
	hipMemcpy(d_inputImage, image, imageSize, hipMemcpyHostToDevice);

	dim3 blockDim(16, 16);
	dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

	// CUDA event timing
	hipEvent_t start, stop;
	float milliseconds = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	// Launch kernel
	d_applyConvolutionKernel << <gridDim, blockDim >> > (d_inputImage, d_outputImage, width, height, kernelSize);

	hipEventRecord(stop);
	hipDeviceSynchronize();
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Kernel execution time: %.6f second\n", (milliseconds / 1000.0f));

	// Copy result back
	unsigned char* outputImage = (unsigned char*)malloc(imageSize);
	hipMemcpy(outputImage, d_outputImage, imageSize, hipMemcpyDeviceToHost);
	save_bmp("lenaout.bmp", outputImage);

	// Free memory
	free(image);
	free(outputImage);
	hipFree(d_inputImage);
	hipFree(d_outputImage);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
